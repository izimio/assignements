// %%writefile cuda_ray.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define SPHERES 20
#define INF 2e10f
#define DIM 2048
#define rnd(x) (x * rand() / RAND_MAX)

struct Sphere
{
    float r, g, b;
    float radius;
    float x, y, z;

    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius)
        {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void kernel(Sphere *s, unsigned char *ptr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= DIM || y >= DIM)
        return;

    int offset = x + y * DIM;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;

    for (int i = 0; i < SPHERES; i++)
    {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz)
        {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char *bitmap, int xdim, int ydim, const char *filename)
{
    FILE *fp = fopen(filename, "w");
    if (!fp)
    {
        fprintf(stderr, "Failed to write file.\n");
        return;
    }
    fprintf(fp, "P3\n%d %d\n255\n", xdim, ydim);
    for (int y = 0; y < ydim; y++)
    {
        for (int x = 0; x < xdim; x++)
        {
            int i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
}

int main()
{
    Sphere h_spheres[SPHERES];
    srand(time(NULL));
    for (int i = 0; i < SPHERES; i++)
    {
        h_spheres[i].r = rnd(1.0f);
        h_spheres[i].g = rnd(1.0f);
        h_spheres[i].b = rnd(1.0f);
        h_spheres[i].x = rnd(2000.0f) - 1000;
        h_spheres[i].y = rnd(2000.0f) - 1000;
        h_spheres[i].z = rnd(2000.0f) - 1000;
        h_spheres[i].radius = rnd(200.0f) + 40;
    }

    Sphere *d_spheres;
    unsigned char *d_bitmap;
    unsigned char *h_bitmap = (unsigned char *)malloc(DIM * DIM * 4);

    hipMalloc(&d_spheres, sizeof(Sphere) * SPHERES);
    hipMemcpy(d_spheres, h_spheres, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

    hipMalloc(&d_bitmap, DIM * DIM * 4);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((DIM + 15) / 16, (DIM + 15) / 16);

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    kernel<<<numBlocks, threadsPerBlock>>>(d_spheres, d_bitmap);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back
    hipMemcpy(h_bitmap, d_bitmap, DIM * DIM * 4, hipMemcpyDeviceToHost);
    ppm_write(h_bitmap, DIM, DIM, "result.ppm");

    printf("CUDA ray tracing: %.3f sec\n", milliseconds / 1000.0f);
    printf("[result.ppm] was generated.\n");

    // Cleanup
    free(h_bitmap);
    hipFree(d_bitmap);
    hipFree(d_spheres);

    return 0;
}
